#include "hip/hip_runtime.h"
#include "kmeans_cuda_basic.h"
#include "random.h"
#include "io.h"
// #include <cmath>
#include <limits>
#include <chrono>
#include <math.h>


void kmeans_cuda_basic(double *dataset, double * centroids, options_t &args) {
  int iterations = 0;
  double * old_centroids = NULL;
  bool done = false;
  int * labels;
  double duration_total = 0;
  double duration = 0;

  while(!done){
    //copy
    duration = 0;

    old_centroids = cuda_copy(centroids, args);

    iterations++;

    //labels is a mapping from each point in the dataset to the enarest euclidian distance centroid
    labels = cuda_find_nearest_centroids(dataset, centroids, args, &duration);

    // Print Labels
    // for (int i =0 ; i< args.number_of_values; i++){
    //   std::cout << i << ": " << labels[i] << std::endl;
    // }

    //the new centroids are the average of all points that map to each centroid
    centroids = cuda_average_labeled_centroids(dataset, labels, args, &duration);

    done = iterations > args.max_num_iter || cuda_converged(centroids, old_centroids, args, &duration);

    duration_total += duration;
    free(old_centroids);
    // free labels, only if not done
    if (!done) free (labels);
  }

  printf("%d,%lf\n", iterations, duration_total/iterations);

  args.labels = labels;
  args.centroids = centroids;
}

int * cuda_find_nearest_centroids(double * h_dataset, double * h_centroids, options_t &args, double * duration){
  //Timing
  hipEvent_t start, stop;
  hipEventCreate(&start);
  hipEventCreate(&stop);

  int * h_labels = (int *)malloc(args.number_of_values * sizeof(int));

  //Allocate Device Memory
  double * d_dataset;
  double * d_centroids;
  double * d_intermediate_values;
  int * d_labels;

  hipMalloc((void**)&d_dataset, args.dims*args.number_of_values*sizeof(double));
  hipMalloc((void**)&d_centroids, args.dims*args.num_cluster*sizeof(double));
  hipMalloc((void**)&d_intermediate_values, args.number_of_values * args.num_cluster * sizeof(double));
  hipMalloc((void**)&d_labels, args.number_of_values * sizeof(int));

  // Transfer Memory from Host to Device
  hipMemcpy(d_dataset, h_dataset, args.dims*args.number_of_values*sizeof(double), hipMemcpyHostToDevice);
  hipMemcpy(d_centroids, h_centroids, args.dims*args.num_cluster*sizeof(double), hipMemcpyHostToDevice);

  //Launch the kernel
  hipEventRecord(start);
  d_cuda_find_nearest_centroids<<<dim3(args.number_of_values), dim3(args.num_cluster)>>>(d_dataset, d_centroids, d_intermediate_values, d_labels, args.dims, std::numeric_limits<double>::max());
  hipEventRecord(stop);

  float ms = 0;
  hipEventElapsedTime(&ms, start, stop);
  *duration += ms;

  //Sync
  hipDeviceSynchronize();

  // Copy Memory back from Device to Host
  hipMemcpy(h_labels, d_labels, args.number_of_values*sizeof(int), hipMemcpyDeviceToHost);

  //Free Device Memory
  hipFree(d_dataset);
  hipFree(d_centroids);
  hipFree(d_intermediate_values);
  hipFree(d_labels);

  return h_labels;
}

__global__ void d_cuda_find_nearest_centroids(double * dataset, double * centroids, double * temp, int * labels, int dims, double max){
  //For the dataset the thread id should not matter as each thread should point to the same point in the dataset, a block maps to a point
  int point_starting_index = blockIdx.x * dims;

  //Likewise each centroid does not care about what block it's in it only cares about what thread it's in, a thread maps to a centroid
  int thread_start_index = threadIdx.x * dims;

  //Unique index we use to store the distance for each point to each centroid
  int stored_index = threadIdx.x + blockIdx.x * blockDim.x;

  if (threadIdx.x < blockDim.x){
    double distance = 0;
    for (int i = 0; i < dims; i++ ){
      // Centroid indexing is different from the indexing of the data set!
      // This needs to be looked into further, when you look at this next write out pseudo code first
      // My thinking was that each <<<block, threads>>> each block would find the label for each point,
      // Where each thread would find the distance for point vs centroid, the block would sync and choose
      // lowest point to assign the label as.
      // In this case the starting index of the centroids is independent of what block you are in
      distance += powf( dataset[point_starting_index+i] - centroids[thread_start_index + i], 2.0);
    }
  // At this point now each thread has caluclated their own distance, this should now be stored somewhere linked to said thread
  // Threads is mapped to the cluster
  distance = sqrtf(distance);
  temp[stored_index] = distance;
  }

  __syncthreads();

  // Each thread at this point now has calculated the distance, we now should find the centroid with the smallest distance.
  if (threadIdx.x == 0) {
    double shortest_distance = max;
    int id_short = 7;
    // Loop through all the clusters, recall that the number of threads is the number of clusters, and here blockDim.x is number of clusters
    for (int j =0; j < blockDim.x; j++){
      if (temp[stored_index + j] < shortest_distance)
      {
        id_short = j;
        shortest_distance = temp[stored_index + j];
      }
    }

    //A label needs to be only stored per point, each block is given a point thus, each block id represents the points index in the label
    labels[blockIdx.x] = id_short;
  }
}
double * cuda_average_labeled_centroids(double * h_dataset, int * h_labels, options_t &args, double * duration){
  //Timing
  hipEvent_t start, stop;
  hipEventCreate(&start);
  hipEventCreate(&stop);

  // First turn the dataset into a singular dimension
  double * h_centroids = (double *)malloc(args.num_cluster * args.dims * sizeof(double));

  // Allocate Device Memory
  double * d_dataset;
  int * d_labels;
  double * d_centroids;
  hipMalloc((void**)&d_dataset, args.number_of_values * args.dims * sizeof(double));
  hipMalloc((void**)&d_labels, args.number_of_values * sizeof(int));
  hipMalloc((void**)&d_centroids, args.num_cluster * args.dims * sizeof(double));

  // Transfer Memory From Host To Device
  hipMemcpy(d_dataset, h_dataset, args.number_of_values * args.dims * sizeof(double), hipMemcpyHostToDevice);
  hipMemcpy(d_labels, h_labels, args.number_of_values * sizeof(int), hipMemcpyHostToDevice);
  hipMemcpy(d_centroids, 0, args.num_cluster * args.dims * sizeof(double), hipMemcpyHostToDevice); // Should start from zero?

  // Launch the kernel
  hipEventRecord(start);
  d_cuda_average_labeled_centroids<<<dim3(args.num_cluster), dim3(args.dims)>>>(d_dataset, d_labels, d_centroids, args.number_of_values);
  hipEventRecord(stop);

  // Sync
  hipDeviceSynchronize();
  // Copy Memory back from Device to Host
  hipMemcpy(h_centroids, d_centroids, args.num_cluster * args.dims * sizeof(double), hipMemcpyDeviceToHost);
  // Free Device Memory
  hipFree(d_dataset);
  hipFree(d_labels);
  hipFree(d_centroids);

  float ms = 0;
  hipEventElapsedTime(&ms, start, stop);
  *duration += ms;
  return h_centroids;
}

__global__ void d_cuda_average_labeled_centroids(double * d_dataset, int * d_labels, double * centroids, int number_of_values){
  // Dimensions is blockDim.x
  // A block here manages the centroid Id
  // A thread here manages the addition it needs to do for that dimension
  int points = 0;
  // First loop through  d_dataset skipping dim[blockDim.x] times, and check if the value here is equal to our block id
  for (int i = 0; i < number_of_values; i ++) {
    if (d_labels[i] == blockIdx.x) {
      points++;
      centroids[blockIdx.x * blockDim.x + threadIdx.x] += d_dataset[i * blockDim.x + threadIdx.x];
    }
  }

  if (points != 0){
    centroids[blockIdx.x * blockDim.x + threadIdx.x] /= points;
  }

  //Once you have done the addition for all

}


bool cuda_converged(double * h_new_centroids, double* h_old_centroids, options_t &args, double * duration) {

  //Timing
  hipEvent_t start, stop;
  hipEventCreate(&start);
  hipEventCreate(&stop);

  bool * h_convergence = (bool *)malloc(args.num_cluster * sizeof(double));

  //Allocate Device Memory
  double * d_new_centroids;
  double * d_old_centroids;
  double * d_intermediate_values;
  bool * d_convergence;

  hipMalloc((void**)&d_new_centroids, args.dims*args.num_cluster*sizeof(double));
  hipMalloc((void**)&d_old_centroids, args.dims*args.num_cluster*sizeof(double));
  hipMalloc((void**)&d_intermediate_values, args.num_cluster*sizeof(double));
  hipMalloc((void**)&d_convergence, args.num_cluster*sizeof(bool));

  // Transfer Memory from Host to Device
  hipMemcpy(d_new_centroids, h_new_centroids, args.dims*args.num_cluster*sizeof(double), hipMemcpyHostToDevice);
  hipMemcpy(d_old_centroids, h_old_centroids, args.dims*args.num_cluster*sizeof(double), hipMemcpyHostToDevice);

  hipEventRecord(start);
  d_cuda_convergence_helper<<<dim3(args.num_cluster), dim3(args.dims)>>>(d_new_centroids, d_old_centroids, d_intermediate_values, d_convergence, args.threshold, args.dims);
  hipEventRecord(stop);

  //Sync
  hipDeviceSynchronize();

  // Copy Memory back from Device to Host
  hipMemcpy(h_convergence, d_convergence, args.num_cluster*sizeof(bool), hipMemcpyDeviceToHost);

  bool converged = true;

  for (int i =0; i < args.num_cluster; i++){
    if (!h_convergence[i]) {
      converged = false;
      break;
    }
  }

  // Free Device Memory
  hipFree(d_new_centroids);
  hipFree(d_old_centroids);
  hipFree(d_intermediate_values);
  hipFree(d_convergence);

  // Free Host Memory
  free(h_convergence);
  float ms = 0;
  hipEventElapsedTime(&ms, start, stop);
  *duration += ms;
  // Check if each of the centroid has moved less than the threshold provided.
  return converged;
}

__global__ void d_cuda_convergence_helper(double * new_c, double * old_c, double * temp, bool * convergence, double threshold, int dimensions){
  int index = threadIdx.x + blockIdx.x * blockDim.x;

  if (threadIdx.x < dimensions){
    atomicAdd(&temp[blockIdx.x], (double)powf( new_c[index] - old_c[index], 2.0));
  }

  __syncthreads();

  // It looks like here maybe we could make use of __atomic_add, would that make a speedup? Not noticeable enough

  if (threadIdx.x == 0) {
    if (threshold < sqrtf(temp[blockIdx.x])){
      convergence[blockIdx.x] = false;
    } else {
      convergence[blockIdx.x] = true;
    }
  }
}

double * cuda_copy(double * original, options_t args)
{
  double * copy = (double *) malloc(args.num_cluster * args.dims * sizeof(double));

  for (int i =0; i < args.num_cluster * args.dims; i++){
    copy[i] = original[i];
  }

  return copy;
}
