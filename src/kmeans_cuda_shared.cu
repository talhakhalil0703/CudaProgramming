#include "hip/hip_runtime.h"
#include "kmeans_cuda_shared.h"
#include "random.h"
#include "io.h"
// #include <cmath>
#include <limits>
#include <chrono>
#include <math.h>

__device__ double shared_atomicMin_d(double* address, double val)
{
    unsigned long long int* address_as_ull = (unsigned long long int*)address;
    unsigned long long int old = *address_as_ull, assumed;
    do {
        assumed = atomicMin(address_as_ull, __double_as_longlong(val));
        old = atomicCAS(address_as_ull, old, assumed);

    // Note: uses integer comparison to avoid hang in case of NaN (since NaN != NaN)
    } while (assumed != old);

    return __longlong_as_double(old);
}

void kmeans_cuda_shared(double * dataset, double * centroids, options_t &args) {

  int iterations = 0;
  double * old_centroids = NULL;
  bool done = false;
  int * labels;
  double duration_total = 0;
  double duration = 0;

  while(!done){
    //copy
    duration = 0;

    old_centroids = cuda_shared_copy(centroids, args);

    iterations++;

    //labels is a mapping from each point in the dataset to the enarest euclidian distance centroid
    labels = cuda_shared_find_nearest_centroids(dataset, centroids, args, &duration);

    // Print Labels
    // for (int i =0 ; i< args.number_of_values; i++){
    //   std::cout << i << ": " << labels[i] << std::endl;
    // }

    //the new centroids are the average of all points that map to each centroid
    centroids = cuda_shared_average_labeled_centroids(dataset, labels, args, &duration);

    done = iterations > args.max_num_iter || cuda_shared_converged(centroids, old_centroids, args, &duration);

    duration_total += duration;
    free(old_centroids);
    // free labels, only if not done
    if (!done) free (labels);
  }

  printf("%d,%lf\n", iterations, duration_total/iterations);

  args.labels = labels;
  args.centroids = centroids;
}

int * cuda_shared_find_nearest_centroids(double * h_dataset, double * h_centroids, options_t &args, double * duration){
  //Timing
  hipEvent_t start, stop;
  hipEventCreate(&start);
  hipEventCreate(&stop);

  int * h_labels = (int *)malloc(args.number_of_values * sizeof(int));

  //Allocate Device Memory
  double * d_dataset;
  double * d_centroids;
  int * d_labels;

  hipMalloc((void**)&d_dataset, args.dims*args.number_of_values*sizeof(double));
  hipMalloc((void**)&d_centroids, args.dims*args.num_cluster*sizeof(double));
  hipMalloc((void**)&d_labels, args.number_of_values * sizeof(int));

  // Transfer Memory from Host to Device
  hipMemcpy(d_dataset, h_dataset, args.dims*args.number_of_values*sizeof(double), hipMemcpyHostToDevice);
  hipMemcpy(d_centroids, h_centroids, args.dims*args.num_cluster*sizeof(double), hipMemcpyHostToDevice);

  //Launch the kernel
  hipEventRecord(start);
  d_cuda_shared_find_nearest_centroids<<<dim3(args.number_of_values), dim3(args.num_cluster)>>>(d_dataset, d_centroids, d_labels, args.dims, std::numeric_limits<double>::max());
  hipEventRecord(stop);

  float ms = 0;
  hipEventElapsedTime(&ms, start, stop);
  *duration += ms;

  //Sync
  hipDeviceSynchronize();

  // Copy Memory back from Device to Host
  hipMemcpy(h_labels, d_labels, args.number_of_values*sizeof(int), hipMemcpyDeviceToHost);

  //Free Device Memory
  hipFree(d_dataset);
  hipFree(d_centroids);
  hipFree(d_labels);

  return h_labels;
}

__global__ void d_cuda_shared_find_nearest_centroids(double * dataset, double * centroids, int * labels, int dims, double max){
  __shared__ double s_distance;
  s_distance = max;


  __syncthreads();

  if (threadIdx.x < blockDim.x){
    double distance = 0;
    for (int i = 0; i < dims; i++ ){
      // Centroid indexing is different from the indexing of the data set!
      // This needs to be looked into further, when you look at this next write out pseudo code first
      // My thinking was that each <<<block, threads>>> each block would find the label for each point,
      // Where each thread would find the distance for point vs centroid, the block would sync and choose
      // lowest point to assign the label as.
      // In this case the starting index of the centroids is independent of what block you are in
      distance += powf( dataset[blockIdx.x * dims + i] - centroids[threadIdx.x * dims + i], 2.0);
    }
    // At this point now each thread has caluclated their own distance, this should now be stored somewhere linked to said thread
    // Threads is mapped to the cluster
    distance = sqrtf(distance);
    shared_atomicMin_d(&s_distance, distance);
    __syncthreads();

    if (distance == s_distance){
      labels[blockIdx.x] = threadIdx.x;
    }
  }
}
double * cuda_shared_average_labeled_centroids(double * h_dataset, int * h_labels, options_t &args, double * duration){
  //Timing
  hipEvent_t start, stop;
  hipEventCreate(&start);
  hipEventCreate(&stop);

  // First turn the dataset into a singular dimension
  double * h_centroids = (double *)malloc(args.num_cluster * args.dims * sizeof(double));

  // Allocate Device Memory
  double * d_dataset;
  int * d_labels;
  double * d_centroids;
  hipMalloc((void**)&d_dataset, args.number_of_values * args.dims * sizeof(double));
  hipMalloc((void**)&d_labels, args.number_of_values * sizeof(int));
  hipMalloc((void**)&d_centroids, args.num_cluster * args.dims * sizeof(double));

  // Transfer Memory From Host To Device
  hipMemcpy(d_dataset, h_dataset, args.number_of_values * args.dims * sizeof(double), hipMemcpyHostToDevice);
  hipMemcpy(d_labels, h_labels, args.number_of_values * sizeof(int), hipMemcpyHostToDevice);
  hipMemcpy(d_centroids, 0, args.num_cluster * args.dims * sizeof(double), hipMemcpyHostToDevice); // Should start from zero?

  // Launch the kernel
  hipEventRecord(start);
  d_cuda_shared_average_labeled_centroids<<<dim3(args.num_cluster), dim3(args.dims)>>>(d_dataset, d_labels, d_centroids, args.number_of_values);
  hipEventRecord(stop);

  // Sync
  hipDeviceSynchronize();
  // Copy Memory back from Device to Host
  hipMemcpy(h_centroids, d_centroids, args.num_cluster * args.dims * sizeof(double), hipMemcpyDeviceToHost);
  // Free Device Memory
  hipFree(d_dataset);
  hipFree(d_labels);
  hipFree(d_centroids);

  float ms = 0;
  hipEventElapsedTime(&ms, start, stop);
  *duration += ms;
  return h_centroids;
}

__global__ void d_cuda_shared_average_labeled_centroids(double * d_dataset, int * d_labels, double * centroids, int number_of_values){
  // Dimensions is blockDim.x
  // A block here manages the centroid Id
  // A thread here manages the addition it needs to do for that dimension
  int points = 0;
  // First loop through  d_dataset skipping dim[blockDim.x] times, and check if the value here is equal to our block id
  for (int i = 0; i < number_of_values; i ++) {
    if (d_labels[i] == blockIdx.x) {
      points++;
      centroids[blockIdx.x * blockDim.x + threadIdx.x] += d_dataset[i * blockDim.x + threadIdx.x];
    }
  }

  if (points != 0){
    centroids[blockIdx.x * blockDim.x + threadIdx.x] /= points;
  }

  //Once you have done the addition for all

}


bool cuda_shared_converged(double * h_new_centroids, double* h_old_centroids, options_t &args, double * duration) {

  //Timing
  hipEvent_t start, stop;
  hipEventCreate(&start);
  hipEventCreate(&stop);

  bool * h_convergence = (bool *)malloc(args.num_cluster * sizeof(double));

  //Allocate Device Memory
  double * d_new_centroids;
  double * d_old_centroids;
  bool * d_convergence;

  hipMalloc((void**)&d_new_centroids, args.dims*args.num_cluster*sizeof(double));
  hipMalloc((void**)&d_old_centroids, args.dims*args.num_cluster*sizeof(double));
  hipMalloc((void**)&d_convergence, args.num_cluster*sizeof(bool));

  // Transfer Memory from Host to Device
  hipMemcpy(d_new_centroids, h_new_centroids, args.dims*args.num_cluster*sizeof(double), hipMemcpyHostToDevice);
  hipMemcpy(d_old_centroids, h_old_centroids, args.dims*args.num_cluster*sizeof(double), hipMemcpyHostToDevice);

  hipEventRecord(start);
  d_cuda_shared_convergence_helper<<<dim3(args.num_cluster), dim3(args.dims)>>>(d_new_centroids, d_old_centroids, d_convergence, args.threshold, args.dims);
  hipEventRecord(stop);

  //Sync
  hipDeviceSynchronize();

  // Copy Memory back from Device to Host
  hipMemcpy(h_convergence, d_convergence, args.num_cluster*sizeof(bool), hipMemcpyDeviceToHost);

  bool converged = true;

  for (int i =0; i < args.num_cluster; i++){
    if (!h_convergence[i]) {
      converged = false;
      break;
    }
  }

  // Free Device Memory
  hipFree(d_new_centroids);
  hipFree(d_old_centroids);
  hipFree(d_convergence);

  // Free Host Memory
  free(h_convergence);
  float ms = 0;
  hipEventElapsedTime(&ms, start, stop);
  *duration += ms;
  // Check if each of the centroid has moved less than the threshold provided.
  return converged;
}

__global__ void d_cuda_shared_convergence_helper(double * new_c, double * old_c, bool * convergence, double threshold, int dimensions){
  int index = threadIdx.x + blockIdx.x * blockDim.x;
  __shared__ double distance;
  distance = 0;

  if (threadIdx.x < dimensions){
    atomicAdd(&distance, (double)powf( new_c[index] - old_c[index], 2.0));
  }

  __syncthreads();

  // It looks like here maybe we could make use of __atomic_add, would that make a speedup? Not noticeable enough

  if (threadIdx.x == 0) {
    if (threshold < sqrtf(distance)){
      convergence[blockIdx.x] = false;
    } else {
      convergence[blockIdx.x] = true;
    }
  }
}

double * cuda_shared_copy(double * original, options_t args)
{
  double * copy = (double *) malloc(args.num_cluster * args.dims * sizeof(double));

  for (int i =0; i < args.num_cluster * args.dims; i++){
    copy[i] = original[i];
  }

  return copy;
}
